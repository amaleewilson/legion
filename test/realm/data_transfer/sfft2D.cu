
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <iostream>

const int NX = 2048;
const int NY = 2048;

const int DEFAULT_FFT_TRIALS = 10000;
const int DEFAULT_META_TRIALS = 10;

const int BATCH_SIZE = 1;

int main(int argc, char **argv) {
    int fft_trials = DEFAULT_FFT_TRIALS;
    int meta_trials = DEFAULT_META_TRIALS;

    printf("[INFO] META trials: %d\n", meta_trials);
    printf("[INFO] FFT trials: %d\n", fft_trials);

    long nx = NX;
    long ny = NX;
    printf("[INFO] NX Length: %ld\n", nx);
    printf("[INFO] NY Length: %ld\n", ny);

    hipfftComplex *h_original_signal, *h_applied_fft_signal;
    hipHostMalloc((void **) &h_original_signal, sizeof(hipfftComplex) * NX * NY, hipHostMallocDefault);
    hipHostMalloc((void **) &h_applied_fft_signal, sizeof(hipfftComplex) * NX * NY, hipHostMallocDefault);

    hipfftComplex *d_original_signal, *d_applied_fft_signal;
    hipMalloc((void **) &d_original_signal, sizeof(hipfftComplex) * NX * NY);
    hipMalloc((void **) &d_applied_fft_signal, sizeof(hipfftComplex) * NX * NY);

    /*
     * generate random signal as original signal
     */
    srand(0); // initialize random seed
    for (int i = 0; i < NX*NY; i++) {
        h_original_signal[i].x = (float)((int)rand() % 10);
        h_original_signal[i].y = 0.0;
    }

//    for (int i = 0; i < NX; ++i){
//      for (int j = 0; j < NY; ++j){
//        std::cout << h_original_signal[i*NX + j].x << " ";
//     }
//      std::cout << std::endl;
//    }

    hipMemcpy(d_original_signal, h_original_signal, sizeof(hipfftComplex) * NX * NY, hipMemcpyHostToDevice);

    hipfftHandle fft_plan;
    //cufftPlan1d(&fft_plan, NX, CUFFT_C2C, BATCH_SIZE);
    hipfftPlan2d(&fft_plan, NX, NY, HIPFFT_C2C);
    
//    int *n = new int[2];
//    n[0] = nx;
//    n[1] = ny;
//
//    int *inembed = new int[2];
//    inembed[0] = nx;
//    inembed[1] = ny;
//
//    int istride = 1;
//    int idist = nx*ny;
//
//    cufftPlanMany(&fft_plan, 2, n, inembed, istride, idist, inembed, istride, idist, CUFFT_C2C, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sum_of_elapsed_times = 0.0;

    printf("[INFO] Run benchmark...\n");
    for (int i = 0; i < meta_trials; i++) {
        hipEventRecord(start, 0);

        for (int j = 0; j < fft_trials; j++) {
            hipfftExecC2C(fft_plan, d_original_signal, d_applied_fft_signal, HIPFFT_FORWARD);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_time_ms;
        hipEventElapsedTime(&elapsed_time_ms, start, stop);

        float elapsed_time_sec = elapsed_time_ms / 1000.0;
        sum_of_elapsed_times += elapsed_time_sec;
        printf("%f sec\n", elapsed_time_sec);
    }


    hipMemcpy(h_applied_fft_signal, d_applied_fft_signal, sizeof(hipfftComplex) * NX * NY, hipMemcpyDeviceToHost);

//    printf("[INFO] computing sum...\n");
//    
//    for (int i = 0; i < NX; ++i){
//      for (int j = 0; j < NY; ++j){
//        std::cout << h_applied_fft_signal[i*NX + j].x << " ";
//     }
//      std::cout << std::endl;
//    }
//    
//    float red = 0;
//    for (int i = 0; i < NX*NY; i++) {
//        red += h_applied_fft_signal[i].x;
//        red -= h_applied_fft_signal[i].y;
//    }
//
//    printf("SUM : %f\n", red);


    printf("[INFO] Finished!\n");
    printf("[INFO] Average: %lf sec\n", sum_of_elapsed_times / meta_trials);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

