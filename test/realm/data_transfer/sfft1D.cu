
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>

const int DEFAULT_SIGNAL_LENGTH = 4096;
const int DEFAULT_FFT_TRIALS = 10000;
const int DEFAULT_META_TRIALS = 10;

const int BATCH_SIZE = 1;

int main(int argc, char **argv) {
    int fft_trials = DEFAULT_FFT_TRIALS;
    int meta_trials = DEFAULT_META_TRIALS;

    printf("[INFO] META trials: %d\n", meta_trials);
    printf("[INFO] FFT trials: %d\n", fft_trials);

    long signal_length = DEFAULT_SIGNAL_LENGTH;
    printf("[INFO] Signal Length: %ld\n", signal_length);

    hipfftComplex *h_original_signal;
    hipHostMalloc((void **) &h_original_signal, sizeof(hipfftComplex) * signal_length, hipHostMallocDefault);

    hipfftComplex *d_original_signal, *d_applied_fft_signal;
    hipMalloc((void **) &d_original_signal, sizeof(hipfftComplex) * signal_length);
    hipMalloc((void **) &d_applied_fft_signal, sizeof(hipfftComplex) * signal_length);

    /*
     * generate random signal as original signal
     */
    srand(0); // initialize random seed
    for (int i = 0; i < signal_length; i++) {
        h_original_signal[i].x = (float)rand() / RAND_MAX;
        h_original_signal[i].y = 0.0;
    }
    hipMemcpy(d_original_signal, h_original_signal, sizeof(hipfftComplex) * signal_length, hipMemcpyHostToDevice);

    hipfftHandle fft_plan;
    hipfftPlan1d(&fft_plan, signal_length, HIPFFT_C2C, BATCH_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sum_of_elapsed_times = 0.0;

    printf("[INFO] Run benchmark...\n");
    for (int i = 0; i < meta_trials; i++) {
        hipEventRecord(start, 0);

        for (int j = 0; j < fft_trials; j++) {
            hipfftExecC2C(fft_plan, d_original_signal, d_applied_fft_signal, HIPFFT_FORWARD);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_time_ms;
        hipEventElapsedTime(&elapsed_time_ms, start, stop);

        float elapsed_time_sec = elapsed_time_ms / 1000.0;
        sum_of_elapsed_times += elapsed_time_sec;
        printf("%f sec\n", elapsed_time_sec);
    }
    printf("[INFO] Finished!\n");
    printf("[INFO] Average: %lf sec\n", sum_of_elapsed_times / meta_trials);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
